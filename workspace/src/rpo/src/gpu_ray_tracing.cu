#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <chrono>
#include <filesystem>

#include <hip/hip_runtime.h>

#include "ros_visualizer.h"


const double PRECISION = 1e-4;

__device__ bool isOccupied(const int* occupancy_grid, int x, int y, int z, int dim_x, int dim_y, int dim_z);

__device__ bool castRayDDAGPU(const int* occupancy_grid, int dim_x, int dim_y, int dim_z, float resolution,
    double grid_origin_x, double grid_origin_y, double grid_origin_z, float ox, float oy, float oz,
    float dx, float dy, float dz, float max_range, int target_x, int target_y, int target_z);

__global__ void visibilityKernel(
    const float* origins,
    const float* targets,
    int* visibility_map,
    const int* occupancy_grid,
    int dim_x, int dim_y, int dim_z,
    float grid_x, float grid_y, float grid_z,
    float resolution, float max_range,
    int num_origins, int num_targets
);




int main(int argc, char** argv)
{
    auto start = std::chrono::high_resolution_clock::now();

    // --- Read input parameters ---
    if (argc != 2)
    {
        std::cerr << "Usage: rosrun rpo RPO <parameter_file>" << std::endl;
        return -1;
    }

    const std::string parameters_file = argv[1];

    rpo::Parameters parameters = rpo::Parameters::loadParameters(parameters_file);



    // --- Read 3D models ---                                                                                  
    std::shared_ptr<ColorOcTree> color_model = nullptr;
    std::shared_ptr<rpo::AugmentedOcTree> augmented_model = nullptr;

    std::ifstream file(parameters.paths.color_model);

    if (file.is_open())
    {
        color_model.reset(dynamic_cast<ColorOcTree*>(AbstractOcTree::read(file)));

        std::cout << "Color octree num leaf nodes: " << color_model->getNumLeafNodes() << std::endl;

        file.close();
    }
    else
    {
        std::cerr << "Could not open color octree file!" << std::endl;
        return -1;
    }

    file.open(parameters.paths.augmented_model);

    if (file.is_open())
    {
        augmented_model.reset(dynamic_cast<rpo::AugmentedOcTree*>(AbstractOcTree::read(file)));

        std::cout << "Augmented octree num leaf nodes: " << augmented_model->getNumLeafNodes() << std::endl;

        file.close();
    }        
    else
    {
        std::cerr << "Could not open augmented octree file!" << std::endl;
        return -1;
    }


    assert(color_model->getNumLeafNodes() < 1'000);
    assert(autmented_model->getNumLeafNodes() < 1'000);



    // --- Visualizer initialization ---
    ros::init(argc, argv, "rpo");

    rpo::ROSVisualizer visualizer(augmented_model, color_model, parameters);



    // --- Preprocessing ---
    visualizer.cutUnderGround();
    visualizer.computeGroundZone();
    visualizer.computeGridElements();
    visualizer.computeRayTargets();
    visualizer.computeGeneralVisibility();



    // --- Generating occupancy grid ---
    float resolution = augmented_model->getResolution();

    std::array<double, 6> boundaries = augmented_model->getBoundaries();

    int dim_x = std::ceil((boundaries[1] - boundaries[0]) / resolution);
    int dim_y = std::ceil((boundaries[3] - boundaries[2]) / resolution);
    int dim_z = std::ceil((boundaries[5] - boundaries[4]) / resolution);

    std::vector<int> occupancy_grid(dim_x * dim_y * dim_z, 0);

    for (rpo::AugmentedOcTree::leaf_iterator it = augmented_model->begin_leafs(), end = augmented_model->end_leafs(); it != end; ++it)
    {
        if (augmented_model->isNodeOccupied(*it))
        {
            octomap::point3d coord = it.getCoordinate();
            int x = static_cast<int>((coord.x() - boundaries[0]) / resolution);
            int y = static_cast<int>((coord.y() - boundaries[2]) / resolution);
            int z = static_cast<int>((coord.z() - boundaries[4]) / resolution);
            occupancy_grid[x + y * dim_x + z * dim_x * dim_y] = 1;
        }
    }

    int counter = 0;

    for (size_t i = 0; i < occupancy_grid.size(); ++i)
    {
        if (occupancy_grid[i] == 1) ++counter;
    }

    std::cout << "Occupancy grid elements: " << counter << "\n";



    // --- Setup origins ---
    KeySet origin_keys = visualizer.getBaseReachableElements();
    std::vector<float> origins;

    for (const auto key : origin_keys)
    {
        octomap::point3d pt = augmented_model->keyToCoord(key, augmented_model->getTreeDepth());

        origins.push_back(pt.x());
        origins.push_back(pt.y());
        origins.push_back(pt.z());
    }

    std::cout << "Origins elements: " << origins.size() << "\n";


    // --- Setup targets ---
    std::vector<octomap::OcTreeKey> grid_elements = visualizer.getGridElements();
    std::vector<double> ray_targets = visualizer.getRayTargets();
    std::vector<float> targets;

    float lamp_x, lamp_y;


    for (size_t i = 0; i < grid_elements.size(); ++i)
    {
        octomap::point3d pt = augmented_model->keyToCoord(grid_elements[i], augmented_model->getTreeDepth());

        lamp_x = pt.x();
        lamp_y = pt.y();

        for (size_t j = 1; j < ray_targets.size() - 1; ++j)
        {
            targets.push_back(pt.x());
            targets.push_back(pt.y());
            targets.push_back(ray_targets[j]);
        }
    }

    std::cout << "Target elements: " << targets.size() << "\n";


    // --- Memory allocation ---
    int* d_grid;
    hipMalloc(&d_grid, occupancy_grid.size() * sizeof(int));
    hipMemcpy(d_grid, occupancy_grid.data(), occupancy_grid.size() * sizeof(int), hipMemcpyHostToDevice);

    float* d_origins;
    hipMalloc(&d_origins, origins.size() * sizeof(float));
    hipMemcpy(d_origins, origins.data(), origins.size() * sizeof(float), hipMemcpyHostToDevice);

    float* d_targets;
    hipMalloc(&d_targets, targets.size() * sizeof(float));
    hipMemcpy(d_targets, targets.data(), targets.size() * sizeof(float), hipMemcpyHostToDevice);

    int num_origins = origins.size() / 3;
    int num_targets = targets.size() / 3;
    std::vector<int> visibility_map(num_origins * num_targets);
    int* d_visibility_map;
    hipMalloc(&d_visibility_map, num_origins * num_targets * sizeof(int));


    // --- Ray tracing kernel
    float max_range = 50.0;
    int threadsPerBlock = 256;
    int blocks = (num_origins * num_targets + threadsPerBlock - 1) / threadsPerBlock;

    visibilityKernel<<<blocks, threadsPerBlock>>>(
        d_origins, d_targets, d_visibility_map, d_grid,
        dim_x, dim_y, dim_z, boundaries[0], boundaries[2], boundaries[4],
        resolution, max_range, num_origins, num_targets   
    );

    hipDeviceSynchronize();

    hipMemcpy(visibility_map.data(), d_visibility_map, num_origins * num_targets * sizeof(int), hipMemcpyDeviceToHost);

    int visible_voxels = 0;

    for (size_t i = 0; i < num_origins; ++i)
    {
        bool visible = false;
        for (size_t j = 0; j < num_targets; ++j)
        {
            if (visibility_map[i * num_targets + j] == 1)
            {
                visible = true;
                break;
            }
        }
        if (visible)
        {
            ++visible_voxels;

            octomap::point3d pt(
                origins[3 * i + 0],
                origins[3 * i + 1],
                origins[3 * i + 2]
            );

            octomap::ColorOcTreeNode* node = color_model->search(pt, color_model->getTreeDepth());

            if (node != nullptr) node->setColor(255, 0, 0);
        } 

        
    }

    std::cout << "Number of visible voxels: " << visible_voxels << std::endl;



    // --- Free memory ---
    hipFree(d_grid);
    hipFree(d_origins);
    hipFree(d_targets);
    hipFree(d_visibility_map);



    // Visualization
    const std::string out_model = "/home/appuser/data/visible.ot";

    // visualizer.placeLamp(lamp_x, lamp_y);

    color_model->write(out_model);

    auto stop = std::chrono::high_resolution_clock::now();
    auto dur = std::chrono::duration_cast<std::chrono::seconds>(stop - start);

    std::cout << "Total duration: " << dur.count() << std::endl;
}



__device__ bool isOccupied(const int* occupancy_grid, int x, int y, int z, int dim_x, int dim_y, int dim_z)
{
    if (x < 0 || y < 0 || z < 0 || x >= dim_x || y >= dim_y || z >= dim_z)
    {
        return false;
    }

    int idx = x + y * dim_x + z * dim_x * dim_y;
    return (occupancy_grid[idx] != 0);
}


__device__ bool castRayDDAGPU(const int* occupancy_grid, int dim_x, int dim_y, int dim_z, float resolution,
    double grid_origin_x, double grid_origin_y, double grid_origin_z, float ox, float oy, float oz,
    float dx, float dy, float dz, float max_range, int target_x, int target_y, int target_z)
{
    int x = static_cast<int>((ox - grid_origin_x) / resolution);
    int y = static_cast<int>((oy - grid_origin_y) / resolution);
    int z = static_cast<int>((oz - grid_origin_z) / resolution);


    float length = std::sqrt(dx * dx + dy * dy + dz * dz);
    dx /= length; dy /= length; dz /= length;

    float epsilon = 1e-6f;

    if (std::abs(dx) < epsilon) dx = (dx >= 0.0f ? epsilon : -epsilon);
    if (std::abs(dy) < epsilon) dy = (dy >= 0.0f ? epsilon : -epsilon);
    if (std::abs(dz) < epsilon) dz = (dz >= 0.0f ? epsilon : -epsilon);


    float voxel_size = resolution;
    
    float t_delta_x = voxel_size / std::abs(dx);
    float t_delta_y = voxel_size / std::abs(dy);
    float t_delta_z = voxel_size / std::abs(dz);

    int step_x = (dx > 0) ? 1 : -1;
    int step_y = (dy > 0) ? 1 : -1;
    int step_z = (dz > 0) ? 1 : -1;

    float voxel_border_x = (x + (step_x > 0 ? 1 : 0)) * voxel_size + grid_origin_x;
    float voxel_border_y = (y + (step_y > 0 ? 1 : 0)) * voxel_size + grid_origin_y;
    float voxel_border_z = (z + (step_z > 0 ? 1 : 0)) * voxel_size + grid_origin_z;

    float t_max_x = (voxel_border_x - ox) / dx;
    float t_max_y = (voxel_border_y - oy) / dy;
    float t_max_z = (voxel_border_z - oz) / dz;
    
    float t = 0.0f;

    while (t < max_range)
    {
        // Lamps are just virtually placed thus any collision means unreachability
        if (x == target_x && y == target_y && z == target_z)
        {
            return true;
        }
        else if (isOccupied(occupancy_grid, x, y, z, dim_x, dim_y, dim_z))
        {
            return false;
        }

        if (t_max_x < t_max_y)
        {
            if (t_max_x < t_max_z)
            {
                x += step_x;
                t = t_max_x;
                t_max_x += t_delta_x;
            }
            else
            {
                z += step_z;
                t = t_max_z;
                t_max_z += t_delta_z;
            }
        }
        else
        {
            if (t_max_y < t_max_z)
            {
                y += step_y;
                t = t_max_y;
                t_max_y += t_delta_y;
            }
            else
            {
                z += step_z;
                t = t_max_z;
                t_max_z += t_delta_z;
            }
        }


        if (x < 0 || y < 0 || z < 0 || x >= dim_x || y >= dim_y || z >= dim_z)
        {
            return false;
        }
    }

    return false;
}


__global__ void visibilityKernel(
    const float* origins,
    const float* targets,
    int* visibility_map,
    const int* occupancy_grid,
    int dim_x, int dim_y, int dim_z,
    float grid_x, float grid_y, float grid_z,
    float resolution, float max_range,
    int num_origins, int num_targets
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_rays = num_origins * num_targets;
    if (idx >= total_rays) return;

    int origin_idx = idx / num_targets;
    int target_idx = idx % num_targets;

    // Origins
    float ox = origins[3 * origin_idx + 0];
    float oy = origins[3 * origin_idx + 1];
    float oz = origins[3 * origin_idx + 2];

    // Targets
    float tx = targets[3 * target_idx + 0];
    float ty = targets[3 * target_idx + 1];
    float tz = targets[3 * target_idx + 2];

    // Directions
    float dx = tx - ox;
    float dy = ty - oy;
    float dz = tz - oz;

    // Target int
    int target_x = (tx - grid_x) / resolution;
    int target_y = (ty - grid_y) / resolution;
    int target_z = (tz - grid_z) / resolution;

    bool hit = false;

    for (int d = 0; d < 3; ++d)
    {
        float sx = ox + (d == 0 ? (dx > 0 ? resolution : -resolution) : 0);
        float sy = oy + (d == 1 ? (dy > 0 ? resolution : -resolution) : 0);
        float sz = oz + (d == 2 ? (dz > 0 ? resolution : -resolution) : 0);

        float dir_x = tx - sx;
        float dir_y = ty - sy;
        float dir_z = tz - sz;

        if (castRayDDAGPU(occupancy_grid, dim_x, dim_y, dim_z, resolution, grid_x, grid_y, grid_z,
            sx, sy, sz, dir_x, dir_y, dir_z, max_range, target_x, target_y, target_z))
        {
            hit = true;
            break;
        } 
    } 

    visibility_map[origin_idx * num_targets + target_idx] = hit ? 1 : 0;
}




