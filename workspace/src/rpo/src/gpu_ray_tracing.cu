#include "hip/hip_runtime.h"
#include "gpu_ray_tracing.cuh"

#include <hip/hip_runtime.h>


__device__ bool isOccupied(const int* occupancy_grid, int x, int y, int z, int dim_x, int dim_y, int dim_z)
{
    if (x < 0 || y < 0 || z < 0 || x >= dim_x || y >= dim_y || z >= dim_z)
    {
        return false;
    }

    int idx = x + y * dim_x + z * dim_x * dim_y;
    return (occupancy_grid[idx] != 0);
}


__device__ bool castRayDDAGPU(const int* occupancy_grid, int dim_x, int dim_y, int dim_z, float resolution,
    double grid_origin_x, double grid_origin_y, double grid_origin_z, float ox, float oy, float oz,
    float dx, float dy, float dz, float max_range, int target_x, int target_y, int target_z)
{
    int x = static_cast<int>((ox - grid_origin_x) / resolution);
    int y = static_cast<int>((oy - grid_origin_y) / resolution);
    int z = static_cast<int>((oz - grid_origin_z) / resolution);


    float length = std::sqrt(dx * dx + dy * dy + dz * dz);
    dx /= length; dy /= length; dz /= length;

    float epsilon = 1e-6f;

    if (std::abs(dx) < epsilon) dx = (dx >= 0.0f ? epsilon : -epsilon);
    if (std::abs(dy) < epsilon) dy = (dy >= 0.0f ? epsilon : -epsilon);
    if (std::abs(dz) < epsilon) dz = (dz >= 0.0f ? epsilon : -epsilon);


    float voxel_size = resolution;
    
    float t_delta_x = voxel_size / std::abs(dx);
    float t_delta_y = voxel_size / std::abs(dy);
    float t_delta_z = voxel_size / std::abs(dz);

    int step_x = (dx > 0) ? 1 : -1;
    int step_y = (dy > 0) ? 1 : -1;
    int step_z = (dz > 0) ? 1 : -1;

    float voxel_border_x = (x + (step_x > 0 ? 1 : 0)) * voxel_size + grid_origin_x;
    float voxel_border_y = (y + (step_y > 0 ? 1 : 0)) * voxel_size + grid_origin_y;
    float voxel_border_z = (z + (step_z > 0 ? 1 : 0)) * voxel_size + grid_origin_z;

    float t_max_x = (voxel_border_x - ox) / dx;
    float t_max_y = (voxel_border_y - oy) / dy;
    float t_max_z = (voxel_border_z - oz) / dz;
    
    float t = 0.0f;

    while (t < max_range)
    {
        // Lamps are just virtually placed thus any collision means unreachability
        if (x == target_x && y == target_y && z == target_z)
        {
            return true;
        }
        else if (isOccupied(occupancy_grid, x, y, z, dim_x, dim_y, dim_z))
        {
            return false;
        }

        if (t_max_x < t_max_y)
        {
            if (t_max_x < t_max_z)
            {
                x += step_x;
                t = t_max_x;
                t_max_x += t_delta_x;
            }
            else
            {
                z += step_z;
                t = t_max_z;
                t_max_z += t_delta_z;
            }
        }
        else
        {
            if (t_max_y < t_max_z)
            {
                y += step_y;
                t = t_max_y;
                t_max_y += t_delta_y;
            }
            else
            {
                z += step_z;
                t = t_max_z;
                t_max_z += t_delta_z;
            }
        }


        if (x < 0 || y < 0 || z < 0 || x >= dim_x || y >= dim_y || z >= dim_z)
        {
            return false;
        }
    }

    return false;
}


__global__ void visibilityKernel(
    const float* origins,
    const float* targets,
    int* visibility_map,
    const int* occupancy_grid,
    int dim_x, int dim_y, int dim_z,
    float grid_x, float grid_y, float grid_z,
    float resolution, float max_range,
    int num_origins, int num_targets
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_rays = num_origins * num_targets;
    if (idx >= total_rays) return;

    int origin_idx = idx / num_targets;
    int target_idx = idx % num_targets;

    // Origins
    float ox = origins[3 * origin_idx + 0];
    float oy = origins[3 * origin_idx + 1];
    float oz = origins[3 * origin_idx + 2];

    // Targets
    float tx = targets[3 * target_idx + 0];
    float ty = targets[3 * target_idx + 1];
    float tz = targets[3 * target_idx + 2];

    // Directions
    float dx = tx - ox;
    float dy = ty - oy;
    float dz = tz - oz;

    // Target int
    int target_x = (tx - grid_x) / resolution;
    int target_y = (ty - grid_y) / resolution;
    int target_z = (tz - grid_z) / resolution;

    bool hit = false;

    for (int d = 0; d < 3; ++d)
    {
        float sx = ox + (d == 0 ? (dx > 0 ? resolution : -resolution) : 0);
        float sy = oy + (d == 1 ? (dy > 0 ? resolution : -resolution) : 0);
        float sz = oz + (d == 2 ? (dz > 0 ? resolution : -resolution) : 0);

        float dir_x = tx - sx;
        float dir_y = ty - sy;
        float dir_z = tz - sz;

        if (castRayDDAGPU(occupancy_grid, dim_x, dim_y, dim_z, resolution, grid_x, grid_y, grid_z,
            sx, sy, sz, dir_x, dir_y, dir_z, max_range, target_x, target_y, target_z))
        {
            hit = true;
            break;
        } 
    } 

    visibility_map[origin_idx * num_targets + target_idx] = hit ? 1 : 0;
}


void runKernel(
    const std::vector<int>& occupancy_grid, 
    const std::vector<float>& origins, 
    const std::vector<float>& targets,
    std::vector<int>& visibility_map, 
    int dim_x, int dim_y, int dim_z,
    float grid_x, float grid_y, float grid_z,
    float resolution, float max_range,
    int num_origins, int num_targets
)
{
    // --- Memory allocation ---
    int* d_grid;
    hipMalloc(&d_grid, occupancy_grid.size() * sizeof(int));
    hipMemcpy(d_grid, occupancy_grid.data(), occupancy_grid.size() * sizeof(int), hipMemcpyHostToDevice);

    float* d_origins;
    hipMalloc(&d_origins, origins.size() * sizeof(float));
    hipMemcpy(d_origins, origins.data(), origins.size() * sizeof(float), hipMemcpyHostToDevice);

    float* d_targets;
    hipMalloc(&d_targets, targets.size() * sizeof(float));
    hipMemcpy(d_targets, targets.data(), targets.size() * sizeof(float), hipMemcpyHostToDevice);

    int* d_visibility_map;
    hipMalloc(&d_visibility_map, num_origins * num_targets * sizeof(int));


    // --- Ray tracing kernel
    int threadsPerBlock = 256;
    int blocks = (num_origins * num_targets + threadsPerBlock - 1) / threadsPerBlock;

    visibilityKernel<<<blocks, threadsPerBlock>>>(
        d_origins, d_targets, d_visibility_map, d_grid,
        dim_x, dim_y, dim_z, grid_x, grid_y, grid_z,
        resolution, max_range, num_origins, num_targets   
    );

    hipDeviceSynchronize();

    hipMemcpy(visibility_map.data(), d_visibility_map, num_origins * num_targets * sizeof(int), hipMemcpyDeviceToHost);

    // --- Free memory ---
    hipFree(d_grid);
    hipFree(d_origins);
    hipFree(d_targets);
    hipFree(d_visibility_map);
}





