#include "hip/hip_runtime.h"
#include <chrono>
#include <fstream>
#include <iostream>
#include <memory>

#include <hip/hip_runtime.h>

#include <octomap/octomap.h>
#include <octomap/ColorOcTree.h>


__device__ bool isOccupied(const int* occupancy_grid, int x, int y, int z, int dim_x, int dim_y, int dim_z);

__device__ bool castRayDDAGPU(const int* occupancy_grid, int dim_x, int dim_y, int dim_z, float resolution,
    double grid_origin_x, double grid_origin_y, double grid_origin_z, float ox, float oy, float oz,
    float dx, float dy, float dz, float max_range, int target_x, int target_y, int target_z);

__global__ void rayTraceKernel(const int* occupancy_grid, int dim_x, int dim_y, int dim_z,
    float resolution, double grid_origin_x, double grid_origin_y, double grid_origin_z,
    float* origins, int num_origins, float max_range, float* targets, int num_targets, int* hit_mask);


int main (int argc, char** argv)
{
    if (argc < 2)
    {
        std::cerr << "Usage: rosrun rpo GPU_DDA <octomap.ot>" << std::endl;
        return 1;
    }

    // Read Octomap
    const std::string filename = argv[1];
    std::unique_ptr<octomap::ColorOcTree> tree = nullptr;
    tree.reset(dynamic_cast<octomap::ColorOcTree*>(octomap::AbstractOcTree::read(filename)));

    // Basic parameters
    octomap::point3d origin(0.025f, 0.025f, 0.325f);
    float resolution = tree->getResolution();
    float max_range = 50.0;

    // Multiple origins
    int n_origins = 24;

    std::vector<float> origins(3 * n_origins);

    for (int i = 0; i < n_origins; ++i)
    {
        origins[3 * i + 0] = origin.x();
        origins[3 * i + 1] = origin.y();
        origins[3 * i + 2] = origin.z() + i * resolution;
    }


    // Creating 1D occupancy grid
    auto grid_start = std::chrono::high_resolution_clock::now();

    int total_voxels = 0;

    double min_x, min_y, min_z;
    double max_x, max_y, max_z;
    tree->getMetricMin(min_x, min_y, min_z);
    tree->getMetricMax(max_x, max_y, max_z);

    int dim_x = std::ceil((max_x - min_x) / resolution);
    int dim_y = std::ceil((max_y - min_y) / resolution);
    int dim_z = std::ceil((max_z - min_z) / resolution);

    std::vector<int> occupancy_grid(dim_x * dim_y * dim_z, 0);

    for (auto it = tree->begin_leafs(), end = tree->end_leafs(); it != end; ++it)
    {
        if (tree->isNodeOccupied(*it))
        {
            ++total_voxels;

            octomap::point3d coord = it.getCoordinate();
            int x = static_cast<int>((coord.x() - min_x) / resolution);
            int y = static_cast<int>((coord.y() - min_y) / resolution);
            int z = static_cast<int>((coord.z() - min_z) / resolution);
            occupancy_grid[x + y * dim_x + z * dim_x * dim_y] = 1;
        }
    }

    auto grid_end = std::chrono::high_resolution_clock::now();
    auto grid_duration = std::chrono::duration_cast<std::chrono::microseconds>(grid_end - grid_start);



    // DDA ray tracing on GPU
    auto gpu_start = std::chrono::high_resolution_clock::now();

    std::vector<float> targets;

    for (auto it = tree->begin_leafs(), end = tree->end_leafs(); it != end; ++it)
    {
        octomap::point3d target = it.getCoordinate();

        targets.push_back(target.x());
        targets.push_back(target.y());
        targets.push_back(target.z());
    }

    int n_targets = targets.size() / 3;
    int total_rays = n_origins * n_targets;

    std::vector<int> gpu_hit_mask(total_rays);


    int* d_grid;
    hipMalloc(&d_grid, occupancy_grid.size() * sizeof(int));
    hipMemcpy(d_grid, occupancy_grid.data(), occupancy_grid.size() * sizeof(int), hipMemcpyHostToDevice);
    

    float* d_origins;
    hipMalloc(&d_origins, 3 * n_origins * sizeof(float));
    hipMemcpy(d_origins, origins.data(), 3 * n_origins * sizeof(float), hipMemcpyHostToDevice);


    float* d_targets;
    hipMalloc(&d_targets, 3 * n_targets * sizeof(float));
    hipMemcpy(d_targets, targets.data(), 3 * n_targets * sizeof(float), hipMemcpyHostToDevice);


    int* d_results;
    hipMalloc(&d_results, total_rays * sizeof(int));

    int threadsPerBlock = 256;
    int blocks = (total_rays + threadsPerBlock - 1) / threadsPerBlock;

    rayTraceKernel<<<blocks, threadsPerBlock>>>(
        d_grid, dim_x, dim_y, dim_z, resolution,
        min_x, min_y, min_z, d_origins, n_origins,
        max_range, d_targets, n_targets, d_results
    );

    hipDeviceSynchronize();

    hipMemcpy(gpu_hit_mask.data(), d_results, total_rays * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_grid);
    hipFree(d_origins);
    hipFree(d_targets);
    hipFree(d_results);

    int gpu_hit_count = 0;

    for (size_t i = 0; i < total_rays; ++i)
    {
        if (gpu_hit_mask[i] == 1) ++gpu_hit_count;
    }

    std::vector<int> target_reached(n_targets, 0);

    for (int origin_idx = 0; origin_idx < n_origins; ++origin_idx)
    {
        for (int target_idx = 0; target_idx < n_targets; ++target_idx)
        {
            int idx = origin_idx * n_targets + target_idx;
            if (gpu_hit_mask[idx] == 1)
            {
                target_reached[target_idx] = 1;
            }
        }
    }

    int reached_count = 0;
    for (int i = 0; i < n_targets; ++i)
    {
        if (target_reached[i] == 1) ++reached_count;
    }
    std::cout << "Targets reached: " << reached_count << "\n";


    auto gpu_end = std::chrono::high_resolution_clock::now();
    auto gpu_duration = std::chrono::duration_cast<std::chrono::microseconds>(gpu_end - gpu_start);

    std::cout << gpu_hit_count << " visible voxels out of " << total_voxels << " are computed in " << (grid_duration.count() + gpu_duration.count()) << " microseconds." << std::endl;
}



__device__ bool isOccupied(const int* occupancy_grid, int x, int y, int z, int dim_x, int dim_y, int dim_z)
{
    if (x < 0 || y < 0 || z < 0 || x >= dim_x || y >= dim_y || z >= dim_z)
    {
        return false;
    }

    int idx = x + y * dim_x + z * dim_x * dim_y;
    return (occupancy_grid[idx] != 0);
}




__device__ bool castRayDDAGPU(
    const int* occupancy_grid,
    int dim_x, int dim_y, int dim_z,
    float resolution,
    double grid_origin_x, double grid_origin_y, double grid_origin_z,
    float ox, float oy, float oz,
    float dx, float dy, float dz,
    float max_range, 
    int target_x, int target_y, int target_z)
{
    int x = static_cast<int>((ox - grid_origin_x) / resolution);
    int y = static_cast<int>((oy - grid_origin_y) / resolution);
    int z = static_cast<int>((oz - grid_origin_z) / resolution);


    float length = std::sqrt(dx * dx + dy * dy + dz * dz);
    dx /= length; dy /= length; dz /= length;

    float voxel_size = resolution;
    
    float t_delta_x = voxel_size / std::abs(dx);
    float t_delta_y = voxel_size / std::abs(dy);
    float t_delta_z = voxel_size / std::abs(dz);

    int step_x = (dx > 0) ? 1 : -1;
    int step_y = (dy > 0) ? 1 : -1;
    int step_z = (dz > 0) ? 1 : -1;

    float voxel_border_x = (x + (step_x > 0 ? 1 : 0)) * voxel_size + grid_origin_x;
    float voxel_border_y = (y + (step_y > 0 ? 1 : 0)) * voxel_size + grid_origin_y;
    float voxel_border_z = (z + (step_z > 0 ? 1 : 0)) * voxel_size + grid_origin_z;

    float t_max_x = (voxel_border_x - ox) / dx;
    float t_max_y = (voxel_border_y - oy) / dy;
    float t_max_z = (voxel_border_z - oz) / dz;
    
    float t = 0.0f;

    while (t < max_range)
    {
        if (isOccupied(occupancy_grid, x, y, z, dim_x, dim_y, dim_z))
        {
            return (x == target_x && y == target_y && z == target_z);
        }

        if (t_max_x < t_max_y)
        {
            if (t_max_x < t_max_z)
            {
                x += step_x;
                t = t_max_x;
                t_max_x += t_delta_x;
            }
            else
            {
                z += step_z;
                t = t_max_z;
                t_max_z += t_delta_z;
            }
        }
        else
        {
            if (t_max_y < t_max_z)
            {
                y += step_y;
                t = t_max_y;
                t_max_y += t_delta_y;
            }
            else
            {
                z += step_z;
                t = t_max_z;
                t_max_z += t_delta_z;
            }
        }


        if (x < 0 || y < 0 || z < 0 || x >= dim_x || y >= dim_y || z >= dim_z)
        {
            return false;
        }
    }

    return false;
}



__global__ void rayTraceKernel(const int* occupancy_grid, int dim_x, int dim_y, int dim_z,
    float resolution, double grid_origin_x, double grid_origin_y, double grid_origin_z,
    float* origins, int num_origins, float max_range, float* targets, int num_targets, int* hit_mask)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_rays = num_origins * num_targets;
    if (idx >= total_rays) return;

    int origin_idx = idx / num_targets;
    int target_idx = idx % num_targets;

    float ox = origins[3 * origin_idx + 0];
    float oy = origins[3 * origin_idx + 1];
    float oz = origins[3 * origin_idx + 2];


    float tx = targets[3 * target_idx + 0];
    float ty = targets[3 * target_idx + 1];
    float tz = targets[3 * target_idx + 2];


    float dx = tx - ox;
    float dy = ty - oy;
    float dz = tz - oz;


    int target_x = (tx - grid_origin_x) / resolution;
    int target_y = (ty - grid_origin_y) / resolution;
    int target_z = (tz - grid_origin_z) / resolution;


    bool hit = castRayDDAGPU(
        occupancy_grid, dim_x, dim_y, dim_z,
        resolution,
        grid_origin_x, grid_origin_y, grid_origin_z,
        ox, oy, oz,
        dx, dy, dz,
        max_range,
        target_x, target_y, target_z);

    hit_mask[idx] = hit ? 1 : 0;
}



